#include "hip/hip_runtime.h"
#include <cv.h>
#include <highgui.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define RED 2
#define GREEN 1
#define BLUE 0

using namespace cv;

// Sequential Code on CPU
void imgConvCPU(unsigned char* imgIn, int row, int col, unsigned int maskWidth, unsigned char* imgOut, char* M) {
	for (int i = 0; i < row; ++i)
	{
		for (int j = 0; j < col; ++j)
		{
			int Pixel = 0;
			int start_r = i - (maskWidth/2);
			int start_c = j - (maskWidth/2);
			for (int k = 0; k < maskWidth; ++k)
			{
				for (int l = 0; l < maskWidth; ++l)
				{
					if((k + start_r) >= 0 && (k + start_r) < row && (l + start_c) >= 0 && (l + start_c) < col)
						Pixel += imgIn[(k + start_r) * col + (l + start_c)] * M[k * maskWidth + l];
				}
			}
			Pixel = Pixel < 0 ? 0 : Pixel;
			Pixel = Pixel > 255 ? 255 : Pixel;
			imgOut[i * col + j] = (unsigned char)Pixel;
		}
	}

}

void serial_host(unsigned char* imgIn, int row, int col, unsigned int maskWidth, unsigned char* imgOut, char* M, double& time) {
	/*******************************HOST********************************/
	clock_t tic = clock();
	imgConvCPU(imgIn,row,col,maskWidth,imgOut,M);
  	clock_t toc = clock();
	time = (double)(toc - tic) / CLOCKS_PER_SEC;
	/*****************************END HOST******************************/
}

void sobel_host(Mat& imgIn, Mat& imgOut, double& time){
	/*******************************HOST********************************/
	clock_t tic = clock();
	Sobel(imgIn,imgOut,CV_8UC1,1,0);
  	clock_t toc = clock();
	time = (double)(toc - tic) / CLOCKS_PER_SEC;
	/*****************************END HOST******************************/
}

int main(int argc, char** argv)
{
	char M[] = {-1,0,1,-2,0,2,-1,0,1};
	unsigned int maskWidth = 3;

	/*
	imgIn: 		Original img (Gray scaled)
	imgOut_1:	Sequential convolution on host
	imgOut_2:	Sobel on host
	imgOut_3:	Sequential convolution on device
	imgOut_4:	Sobel on device
	*/

	unsigned char *imgIn, *imgOut_1, *imgOut_3, *imgOut_4;
	double CPU, CPU_CV, GPU, GPU_CV, acc1, acc2, acc3;
	CPU = CPU_CV = GPU = GPU_CV = acc1 = acc2 = acc3 = 0.0;
	
	// Meaning of  positions: {CPU, CPU_CV, GPU, GPU_CV}
	bool op[] = {false, false, false, false};

	if(argc < 2) {
		printf("No image name given\n");
		return -1;
	}
	char* imageName = argv[1];

	for (int i = 2; i < argc; i++) {
		std::string s = argv[i];
		if (s == "seq_h")
			op[0] = true;
		else if (s == "sobel_h")
			op[1] = true;
		else if (s == "seq_d")
			op[1] = true;
		else if (s == "sobel_d")
			op[2] = true;
	}

	Mat image;
	image = imread(imageName, CV_LOAD_IMAGE_GRAYSCALE);


	// Get image dimension
	Size s = image.size();
	int col = s.width;
	int row = s.height;
	
	int size = sizeof(unsigned char)*row*col;
	int sizeGray = sizeof(unsigned char)*row*col;

	imgIn = (unsigned char*)malloc(size);
	imgOut_1 = (unsigned char*)malloc(sizeGray);
	imgOut_3 = (unsigned char*)malloc(sizeGray);
	imgOut_4 = (unsigned char*)malloc(sizeGray);

	imgIn = image.data;	

	Mat result, imgOut_2;
	imgOut_2.create(row,col,CV_8UC1);

	if (op[0]) serial_host(imgIn, row, col, maskWidth, imgOut_1, M, CPU);
	if (op[1]) sobel_host(image, imgOut_2, CPU_CV);
	// if (op[2]) serial_device(A, B, C3, GPU_tiled, size, N);
	// if (op[3]) sobel_device(A, B, C3, GPU_tiled, size, N);
	
	result.create(row,col,CV_8UC1);

	if (op[0]) {
		printf(" %f |", CPU);
		result.data = imgOut_1;
		imwrite("res_CPU.jpg", result);
	}
	else printf(" - |");

	if (op[1]) {
		if (op[0]) {
			acc1 = CPU / CPU_CV;
			printf(" %f | %f |", CPU_CV, acc1);
		}
		else printf(" %f | - |", CPU_CV);
		imwrite("res_CPU_CV.jpg", imgOut_2);
	}
	else printf(" - | - |");

	if (op[2]) {
		if (op[0]) {
			acc2 = CPU / GPU;
			printf(" %f | %f |", GPU, acc2);
		}
		else printf(" %f | - |", GPU);
	}
	else printf(" - | - |");

	if (op[3]) {
		if (op[0]) {
			acc3 = CPU / GPU_CV;
			printf(" %f | %f |\n", GPU_CV, acc3);
		}
		else printf(" %f | - |\n", GPU_CV);
	}
	else printf(" - | - |\n");


	free(imgOut_1);
	free(imgOut_3);
	free(imgOut_4);
	
	return 0;
}
