#include "hip/hip_runtime.h"
#include <cv.h>
#include <highgui.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math_functions.h>

#define RED 2
#define GREEN 1
#define BLUE 0

using namespace cv;

// Sequential Code on GPU (CUDA)
__global__
void imgConvGPU(unsigned char* imgIn, int row, int col, unsigned int maskWidth, unsigned char* imgOut, char* M) {
	unsigned int row_d = blockIdx.y*blockDim.y+threadIdx.y;
	unsigned int col_d = blockIdx.x*blockDim.x+threadIdx.x;

	int start_r = row_d - (maskWidth/2);
	int start_c = col_d - (maskWidth/2);

	int Pixel = 0;

	for (int k = 0; k < maskWidth; ++k)
	{
		for (int l = 0; l < maskWidth; ++l)
		{
			if((k + start_r) >= 0 && (k + start_r) < row && (l + start_c) >= 0 && (l + start_c) < col)
				Pixel += imgIn[(k + start_r) * col + (l + start_c)] * M[k * maskWidth + l];
		}
	}

	Pixel = Pixel < 0 ? 0 : Pixel;
	Pixel = Pixel > 255 ? 255 : Pixel;
	imgOut[row_d * col + col_d] = (unsigned char)Pixel;
}

// Sequential Code on CPU
void imgConvCPU(unsigned char* imgIn, int row, int col, unsigned int maskWidth, unsigned char* imgOut, char* M) {
	for (int i = 0; i < row; ++i)
	{
		for (int j = 0; j < col; ++j)
		{
			int Pixel = 0;
			int start_r = i - (maskWidth/2);
			int start_c = j - (maskWidth/2);
			for (int k = 0; k < maskWidth; ++k)
			{
				for (int l = 0; l < maskWidth; ++l)
				{
					if((k + start_r) >= 0 && (k + start_r) < row && (l + start_c) >= 0 && (l + start_c) < col)
						Pixel += imgIn[(k + start_r) * col + (l + start_c)] * M[k * maskWidth + l];
				}
			}
			Pixel = Pixel < 0 ? 0 : Pixel;
			Pixel = Pixel > 255 ? 255 : Pixel;
			imgOut[i * col + j] = (unsigned char)Pixel;
		}
	}

}

void serial_host(unsigned char* imgIn, int row, int col, unsigned int maskWidth, unsigned char* imgOut, char* M, double& time) {
	/*******************************HOST********************************/
	clock_t tic = clock();
	imgConvCPU(imgIn,row,col,maskWidth,imgOut,M);
  	clock_t toc = clock();
	time = (double)(toc - tic) / CLOCKS_PER_SEC;
	/*****************************END HOST******************************/
}

void sobel_host(Mat& imgIn, Mat& imgOut, double& time){
	/*******************************HOST********************************/
	clock_t tic = clock();
	Sobel(imgIn,imgOut,CV_8UC1,1,0);
  	clock_t toc = clock();
	time = (double)(toc - tic) / CLOCKS_PER_SEC;
	/*****************************END HOST******************************/
}

void checkError(hipError_t error, std::string type) {
	if(error != hipSuccess){
		printf("Error in %s\n", type.c_str());
		exit(0);
	}
}

void serial_device(unsigned char* imgIn, int row, int col, unsigned int maskWidth, unsigned char* imgOut, char* M, int size, double& time) {
	int size_M = sizeof(unsigned char)*9;
	hipError_t error = hipSuccess;
	unsigned char *d_dataRawImage, *d_imageOutput;
	char* d_M;

	error = hipMalloc((void**)&d_dataRawImage,size);
	checkError(error, "hipMalloc for d_dataRawImage (cuda)");

	error = hipMalloc((void**)&d_imageOutput,size);
	checkError(error, "hipMalloc for d_imageOutput (cuda)");

	error = hipMalloc((void**)&d_M,size_M);
	checkError(error, "hipMalloc for d_M (cuda)");

	/*******************************GPU********************************/
	clock_t tic = clock();

	error = hipMemcpy(d_dataRawImage,imgIn,size,hipMemcpyHostToDevice);
	checkError(error, "hipMemcpy for d_dataRawImage (cuda)");

	error = hipMemcpy(d_M,M,size_M,hipMemcpyHostToDevice);
	checkError(error, "hipMemcpy for d_M (cuda)");
		
	dim3 dimBlock(32,32);
	dim3 dimGrid(ceil(col/float(dimBlock.x)),ceil(row/float(dimBlock.y)));

	imgConvGPU<<<dimGrid,dimBlock>>>(d_dataRawImage, row, col, maskWidth, d_imageOutput, d_M);
	hipDeviceSynchronize();

	error = hipMemcpy(imgOut,d_imageOutput,size,hipMemcpyDeviceToHost);
	checkError(error, "hipMemcpy for imgOut (cuda)");

	clock_t toc = clock();
	time = (double)(toc - tic) / CLOCKS_PER_SEC;
	/*****************************GPU END******************************/

	hipFree(d_dataRawImage);
	hipFree(d_imageOutput);
	hipFree(d_M);
}

int main(int argc, char** argv)
{
	char M[] = {-1,0,1,-2,0,2,-1,0,1};
	unsigned int maskWidth = 3;

	/*
	imgIn: 		Original img (Gray scaled)
	imgOut_1:	Sequential convolution on host
	imgOut_2:	Sobel on host
	imgOut_3:	Sequential convolution on device
	imgOut_4:	Sobel on device
	*/

	unsigned char *imgIn, *imgOut_1, *imgOut_3, *imgOut_4;
	double CPU, CPU_CV, GPU, GPU_CV, acc1, acc2, acc3;
	CPU = CPU_CV = GPU = GPU_CV = acc1 = acc2 = acc3 = 0.0;
	
	// Meaning of  positions: {CPU, CPU_CV, GPU, GPU_CV}
	bool op[] = {false, false, false, false};

	if(argc < 2) {
		printf("No image name given\n");
		return -1;
	}
	char* imageName = argv[1];

	for (int i = 2; i < argc; i++) {
		std::string s = argv[i];
		if (s == "seq_h")
			op[0] = true;
		else if (s == "sobel_h")
			op[1] = true;
		else if (s == "seq_d")
			op[2] = true;
		else if (s == "sobel_d")
			op[3] = true;
	}

	Mat image;
	image = imread(imageName, CV_LOAD_IMAGE_GRAYSCALE);


	// Get image dimension
	Size s = image.size();
	int col = s.width;
	int row = s.height;
	
	int size = sizeof(unsigned char)*row*col;
	int sizeGray = sizeof(unsigned char)*row*col;

	imgIn = (unsigned char*)malloc(size);
	imgOut_1 = (unsigned char*)malloc(sizeGray);
	imgOut_3 = (unsigned char*)malloc(sizeGray);
	imgOut_4 = (unsigned char*)malloc(sizeGray);

	imgIn = image.data;	

	Mat result, imgOut_2;
	imgOut_2.create(row,col,CV_8UC1);

	if (op[0]) serial_host(imgIn, row, col, maskWidth, imgOut_1, M, CPU);
	if (op[1]) sobel_host(image, imgOut_2, CPU_CV);
	if (op[2]) serial_device(imgIn, row, col, maskWidth, imgOut_3, M, sizeGray, GPU);
	// if (op[3]) sobel_device(A, B, C3, GPU_tiled, size, N);
	
	result.create(row,col,CV_8UC1);

	if (op[0]) {
		printf(" %f |", CPU);
		result.data = imgOut_1;
		imwrite("res_CPU.jpg", result);
	}
	else printf(" - |");

	if (op[1]) {
		if (op[0]) {
			acc1 = CPU / CPU_CV;
			printf(" %f | %f |", CPU_CV, acc1);
		}
		else printf(" %f | - |", CPU_CV);
		imwrite("res_CPU_CV.jpg", imgOut_2);
	}
	else printf(" - | - |");

	if (op[2]) {
		if (op[0]) {
			acc2 = CPU / GPU;
			printf(" %f | %f |", GPU, acc2);
		}
		else printf(" %f | - |", GPU);
		result.data = imgOut_3;
		imwrite("res_GPU.jpg", result);
	}
	else printf(" - | - |");

	if (op[3]) {
		if (op[0]) {
			acc3 = CPU / GPU_CV;
			printf(" %f | %f |\n", GPU_CV, acc3);
		}
		else printf(" %f | - |\n", GPU_CV);
	}
	else printf(" - | - |\n");


	free(imgOut_1);
	free(imgOut_3);
	free(imgOut_4);
	
	return 0;
}
