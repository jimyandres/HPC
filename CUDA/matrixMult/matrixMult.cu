#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>


__global__ void matrixMultGPU (double *A, double *B, double *C, int N){
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	double acc;
	if(col < N && row < N){
		acc = 0.0;
		for(int k=0;k<N;k++)
			acc += A[row*N+k] * B[k*N+col];
		C[row*N+col] = acc;
	}
}

void matrixMultCPU(double *A, double *B, double *C, int N){
	double acc;
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			acc=0.0;
			for(int k=0;k<N;k++)
				acc += A[i*N+k]*B[k*N+j];
			C[i*N+j] = acc;
		}
	}
}

int main(int argc, char **argv){
	hipError_t error = hipSuccess;	
	double *A, *B, *C1, *C2;
	double *d_A, *d_B, *d_C;
	double CPU, GPU;
	if(argc != 2) {
		printf("No size given\n");
		return -1;
	}
	int N = atoi(argv[1]);

	double size = N*N*sizeof(double);

  	A = (double*)malloc(size);
 	B = (double*)malloc(size);
 	C1 = (double*)malloc(size);
 	C2 = (double*)malloc(size);

	for(int i=0;i<N*N;i++){
			A[i]=1;
			B[i]=2;
	}

	//CPU----------------------------
	clock_t tic = clock();
	matrixMultCPU(A,B,C1, N);
  	clock_t toc = clock();
	//printf("Tiempo CPU: %f segundos", (double)(toc - tic) / CLOCKS_PER_SEC);
	CPU = (double)(toc - tic) / CLOCKS_PER_SEC;
	printf("%f,",CPU);
	//-------------------------------
  
	error = hipMalloc((void**)&d_A,size);
	if(error != hipSuccess){
		printf("Error in hipMalloc for d_A\n");
		exit(0);
	}
	
	error = hipMalloc((void**)&d_B,size);
	if(error != hipSuccess){
                printf("Error in hipMalloc for d_B\n");
                exit(0);
        }

	error = hipMalloc((void**)&d_C,size);
	if(error != hipSuccess){
                printf("Error in hipMalloc for d_C\n");
                exit(0);
        }

	error = hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
	if(error != hipSuccess){
                printf("Error in hipMemcpy for d_A\n");
                exit(0);
        }

	error = hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
	if(error != hipSuccess){
                printf("Error in hipMemcpy for d_B\n");
                exit(0);
        }

	//GPU----------------------------
	dim3 dimBlock(32,32);
  	dim3 dimGrid(ceil(N/(dimBlock.x)),ceil(N/(dimBlock.y)));
	
  	clock_t tic2 = clock();
	matrixMultGPU<<<dimGrid,dimBlock>>>(d_A,d_B,d_C,N);
  	//cudaDeviceSynchronize();
	hipMemcpy(C2,d_C,size,hipMemcpyDeviceToHost);
  	clock_t toc2 = clock();
	//printf("\n\nTiempo GPU: %f segundos\n", (double)(toc2 - tic2) / CLOCKS_PER_SEC);
	GPU = (double)(toc2 - tic2) / CLOCKS_PER_SEC;
	printf("%f,%f\n",GPU,(CPU/GPU));
	//--------------------------------
  
  	/*for(int i=0;i<N*N;i++){
		if(i%N == 0)
		printf("\n");
			printf("%d ;",A[i]);
	}
	printf("\n---------\n");
	
	for(int i=0;i<N*N;i++){
		if(i%N == 0)
		printf("\n");
			printf("%d ;",B[i]);
	}
	printf("\n---------\n");
	for(int i=0;i<N*N;i++){
		if(i%N == 0)
		printf("\n");
			printf("%d ;",C[i]);
	}
	printf("\n---------\n");*/


	free(A);
	free(B);
	free(C1);
	free(C2);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	
	return 0;
}
